
#include <hip/hip_runtime.h>
typedef struct _MotionThresholds
{
    float motion_threshold_squared;
    float magnitude_quadrant_threshold_squared;
} MotionThresholds;

typedef struct _MotionFeatures
{
    unsigned int pixels;
    unsigned int count;
    float x0_to_x1_magnitude;
    float x1_to_x0_magnitude;
    float y0_to_y1_magnitude;
    float y1_to_y0_magnitude;
} MotionFeatures;

typedef struct _FrameDimensions
{
    size_t width;
    size_t height;
} FrameDimensions;

typedef struct _CUDA2DPitchedArray
{
    void *device_ptr;
    size_t pitch;
    size_t width;
    size_t height;
    size_t elem_size;
} CUDA2DPitchedArray;

extern "C" __global__ void gst_cuda_feature_extractor_kernel(
    const CUDA2DPitchedArray flow_vector_matrix,
    const FrameDimensions frame_dimensions,
    const int flow_vector_grid_size,
    const MotionThresholds flow_vector_thresholds,
    CUDA2DPitchedArray flow_features_matrix)
{
    unsigned int y_frame_idx = (((blockIdx.y * blockDim.y) + threadIdx.y));
    unsigned int x_frame_idx = (((blockIdx.x * blockDim.x) + threadIdx.x));
    unsigned int y_idx = (y_frame_idx / flow_vector_grid_size);
    unsigned int x_idx = (x_frame_idx / flow_vector_grid_size);

    __shared__ unsigned int block_count;
    __shared__ unsigned int block_pixels;
    __shared__ float block_x0_to_x1_magnitude;
    __shared__ float block_x1_to_x0_magnitude;
    __shared__ float block_y0_to_y1_magnitude;
    __shared__ float block_y1_to_y0_magnitude;

    atomicExch(&block_count, 0);
    atomicExch(&block_pixels, 0);
    atomicExch(&block_x0_to_x1_magnitude, 0.0f);
    atomicExch(&block_x1_to_x0_magnitude, 0.0f);
    atomicExch(&block_y0_to_y1_magnitude, 0.0f);
    atomicExch(&block_y1_to_y0_magnitude, 0.0f);

    __syncthreads();

    if(y_frame_idx < frame_dimensions.height
       && x_frame_idx < frame_dimensions.width
       && y_idx < flow_vector_matrix.height
       && x_idx < (flow_vector_matrix.width / flow_vector_matrix.elem_size))
    {
        unsigned int y_offset_index = y_idx * flow_vector_matrix.pitch;

        float flow_vector_x = 0.0;
        float flow_vector_y = 0.0;

        switch(flow_vector_matrix.elem_size)
        {
            case sizeof(short2):
                {
                    short2 *flow_vectors
                        = (short2
                               *)((char *)(flow_vector_matrix.device_ptr) + y_offset_index)
                          + x_idx;

                    flow_vector_x = (float)(flow_vectors->x / (float)(1 << 5));
                    flow_vector_y = (float)(flow_vectors->y / (float)(1 << 5));
                }
                break;
            case sizeof(float2):
                {
                    float2 *flow_vectors
                        = (float2
                               *)((char *)(flow_vector_matrix.device_ptr) + y_offset_index)
                          + x_idx;
                    flow_vector_x = flow_vectors->x;
                    flow_vector_y = flow_vectors->y;
                }
                break;
            default:
                break;
        }

        float flow_vector_x_squared = flow_vector_x * flow_vector_x;
        float flow_vector_y_squared = flow_vector_y * flow_vector_y;
        float distance_squared
            = (flow_vector_x_squared + flow_vector_y_squared);

        atomicAdd((unsigned int *)&block_pixels, 1);

        if(flow_vector_x_squared
           > flow_vector_thresholds.magnitude_quadrant_threshold_squared)
        {
            if(flow_vector_x >= 0)
            {
                atomicAdd((float *)&block_x0_to_x1_magnitude, flow_vector_x);
            }
            else
            {
                atomicAdd((float *)&block_x1_to_x0_magnitude, -flow_vector_x);
            }
        }

        if(flow_vector_y_squared
           > flow_vector_thresholds.magnitude_quadrant_threshold_squared)
        {
            if(flow_vector_y >= 0)
            {
                atomicAdd((float *)&block_y0_to_y1_magnitude, flow_vector_y);
            }
            else
            {
                atomicAdd((float *)&block_y1_to_y0_magnitude, -flow_vector_y);
            }
        }

        if(distance_squared > flow_vector_thresholds.motion_threshold_squared)
        {
            atomicAdd((unsigned int *)&block_count, 1);
        }
    }

    __syncthreads();

    if(threadIdx.y + 1 == blockDim.y && threadIdx.x + 1 == blockDim.x)
    {
        unsigned int y_block_offset_index
            = blockIdx.y * flow_features_matrix.pitch;
        MotionFeatures *flow_features
            = (MotionFeatures
                   *)((char *)(flow_features_matrix.device_ptr) + y_block_offset_index)
              + blockIdx.x;
        flow_features->pixels = block_pixels;
        flow_features->count = block_count;
        flow_features->x0_to_x1_magnitude = block_x0_to_x1_magnitude;
        flow_features->x1_to_x0_magnitude = block_x1_to_x0_magnitude;
        flow_features->y0_to_y1_magnitude = block_y0_to_y1_magnitude;
        flow_features->y1_to_y0_magnitude = block_y1_to_y0_magnitude;
    }
}

extern "C" __global__ void gst_cuda_feature_consolidation_kernel(
    const CUDA2DPitchedArray flow_features_matrix,
    CUDA2DPitchedArray consolidated_flow_features_matrix)
{
    unsigned int y_idx = ((blockIdx.y * blockDim.y) + threadIdx.y);
    unsigned int x_idx = ((blockIdx.x * blockDim.x) + threadIdx.x);

    __shared__ unsigned int consolidated_block_count;
    __shared__ unsigned int consolidated_block_pixels;
    __shared__ float consolidated_block_x0_to_x1_magnitude;
    __shared__ float consolidated_block_x1_to_x0_magnitude;
    __shared__ float consolidated_block_y0_to_y1_magnitude;
    __shared__ float consolidated_block_y1_to_y0_magnitude;

    atomicExch(&consolidated_block_count, 0);
    atomicExch(&consolidated_block_pixels, 0);
    atomicExch(&consolidated_block_x0_to_x1_magnitude, 0.0f);
    atomicExch(&consolidated_block_x1_to_x0_magnitude, 0.0f);
    atomicExch(&consolidated_block_y0_to_y1_magnitude, 0.0f);
    atomicExch(&consolidated_block_y1_to_y0_magnitude, 0.0f);

    __syncthreads();

    if(y_idx < flow_features_matrix.height
       && x_idx < (flow_features_matrix.width / sizeof(MotionFeatures)))
    {
        unsigned int y_offset_index = y_idx * flow_features_matrix.pitch;
        MotionFeatures *original_flow_features
            = ((MotionFeatures
                    *)((char *)(flow_features_matrix.device_ptr) + y_offset_index)
               + x_idx);

        atomicAdd(
            (unsigned int *)(&consolidated_block_pixels),
            original_flow_features->pixels);
        atomicAdd(
            (unsigned int *)(&consolidated_block_count),
            original_flow_features->count);
        atomicAdd(
            (float *)(&consolidated_block_x0_to_x1_magnitude),
            original_flow_features->x0_to_x1_magnitude);
        atomicAdd(
            (float *)(&consolidated_block_x1_to_x0_magnitude),
            original_flow_features->x1_to_x0_magnitude);
        atomicAdd(
            (float *)(&consolidated_block_y0_to_y1_magnitude),
            original_flow_features->y0_to_y1_magnitude);
        atomicAdd(
            (float *)(&consolidated_block_y1_to_y0_magnitude),
            original_flow_features->y1_to_y0_magnitude);
    }

    __syncthreads();

    if(threadIdx.y + 1 == blockDim.y && threadIdx.x + 1 == blockDim.x)
    {

        unsigned int y_block_offset_index
            = blockIdx.y * consolidated_flow_features_matrix.pitch;
        MotionFeatures *flow_features
            = (MotionFeatures
                   *)((char *)(consolidated_flow_features_matrix.device_ptr) + y_block_offset_index)
              + blockIdx.x;
        flow_features->pixels = consolidated_block_pixels;
        flow_features->count = consolidated_block_count;
        flow_features->x0_to_x1_magnitude
            = consolidated_block_x0_to_x1_magnitude;
        flow_features->x1_to_x0_magnitude
            = consolidated_block_x1_to_x0_magnitude;
        flow_features->y0_to_y1_magnitude
            = consolidated_block_y0_to_y1_magnitude;
        flow_features->y1_to_y0_magnitude
            = consolidated_block_y1_to_y0_magnitude;
    }
}
