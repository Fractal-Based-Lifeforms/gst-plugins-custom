
#include <hip/hip_runtime.h>
typedef struct _CUDA2DPitchedArray
{
    void *device_ptr;
    size_t pitch;
    size_t width;
    size_t height;
    size_t elem_size;
} CUDA2DPitchedArray;

typedef struct _FrameDimensions
{
    size_t width;
    size_t height;
} FrameDimensions;

extern "C" __global__ void gst_cuda_feature_extractor_kernel(
    const CUDA2DPitchedArray flow_vector_matrix,
    const FrameDimensions frame_dimensions,
    const int flow_vector_grid_size,
    const float flow_vector_threshold,
    CUDA2DPitchedArray flow_features_matrix)
{
    unsigned int y_frame_idx = (((blockIdx.y * blockDim.y) + threadIdx.y));
    unsigned int x_frame_idx = (((blockIdx.x * blockDim.x) + threadIdx.x));
    unsigned int y_idx = (y_frame_idx / flow_vector_grid_size);
    unsigned int x_idx = (x_frame_idx / flow_vector_grid_size);

    __shared__ float block_spatial_magnitude;

    atomicExch(&block_spatial_magnitude, 0.0f);

    __syncthreads();

    if(y_frame_idx < frame_dimensions.height
       && x_frame_idx < frame_dimensions.width
       && y_idx < flow_vector_matrix.height
       && x_idx < (flow_vector_matrix.width / flow_vector_matrix.elem_size))
    {
        unsigned int y_offset_index = y_idx * flow_vector_matrix.pitch;

        float flow_vector_x = 0.0;
        float flow_vector_y = 0.0;

        switch(flow_vector_matrix.elem_size)
        {
            case sizeof(short2):
                {
                    short2 *flow_vectors
                        = (short2
                               *)((char *)(flow_vector_matrix.device_ptr) + y_offset_index)
                          + x_idx;

                    flow_vector_x = (float)(flow_vectors->x / (float)(1 << 5));
                    flow_vector_y = (float)(flow_vectors->y / (float)(1 << 5));
                }
                break;
            case sizeof(float2):
                {
                    float2 *flow_vectors
                        = (float2
                               *)((char *)(flow_vector_matrix.device_ptr) + y_offset_index)
                          + x_idx;
                    flow_vector_x = flow_vectors->x;
                    flow_vector_y = flow_vectors->y;
                }
                break;
            default:
                break;
        }

        float flow_vector_x_squared = flow_vector_x * flow_vector_x;
        float flow_vector_y_squared = flow_vector_y * flow_vector_y;

        if(flow_vector_x_squared > flow_vector_threshold)
        {
            if(flow_vector_x >= 0)
            {
                atomicAdd((float *)&block_spatial_magnitude, flow_vector_x);
            }
            else
            {
                atomicAdd((float *)&block_spatial_magnitude, -flow_vector_x);
            }
        }

        if(flow_vector_y_squared > flow_vector_threshold)
        {
            if(flow_vector_y >= 0)
            {
                atomicAdd((float *)&block_spatial_magnitude, flow_vector_y);
            }
            else
            {
                atomicAdd((float *)&block_spatial_magnitude, -flow_vector_y);
            }
        }
    }

    __syncthreads();

    if(threadIdx.y + 1 == blockDim.y && threadIdx.x + 1 == blockDim.x)
    {
        unsigned int y_block_offset_index
            = blockIdx.y * flow_features_matrix.pitch;
        float *flow_spatial_feature
            = (float
                   *)((char *)(flow_features_matrix.device_ptr) + y_block_offset_index)
              + blockIdx.x;
        *flow_spatial_feature = block_spatial_magnitude;
    }
}

extern "C" __global__ void gst_cuda_feature_consolidation_kernel(
    const CUDA2DPitchedArray flow_spatial_feature_matrix,
    CUDA2DPitchedArray consolidated_flow_spatial_feature_matrix)
{
    unsigned int y_idx = ((blockIdx.y * blockDim.y) + threadIdx.y);
    unsigned int x_idx = ((blockIdx.x * blockDim.x) + threadIdx.x);

    __shared__ float consolidated_block_spatial_magnitude;

    atomicExch(&consolidated_block_spatial_magnitude, 0.0f);

    __syncthreads();

    if(y_idx < flow_spatial_feature_matrix.height
       && x_idx < (flow_spatial_feature_matrix.width / sizeof(float)))
    {
        unsigned int y_offset_index = y_idx * flow_spatial_feature_matrix.pitch;
        float *original_flow_spatial_feature
            = ((float
                    *)((char *)(flow_spatial_feature_matrix.device_ptr) + y_offset_index)
               + x_idx);

        atomicAdd(
            (float *)(&consolidated_block_spatial_magnitude),
            *original_flow_spatial_feature);
    }

    __syncthreads();

    if(threadIdx.y + 1 == blockDim.y && threadIdx.x + 1 == blockDim.x)
    {

        unsigned int y_block_offset_index
            = blockIdx.y * consolidated_flow_spatial_feature_matrix.pitch;
        float *flow_spatial_feature
            = (float
                   *)((char *)(consolidated_flow_spatial_feature_matrix.device_ptr) + y_block_offset_index)
              + blockIdx.x;
        *flow_spatial_feature = consolidated_block_spatial_magnitude;
    }
}
